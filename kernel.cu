
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel that runs on the GPU
__global__ void helloCUDA() {
    printf("Hello from CUDA kernel! Block: %d, Thread: %d\n", blockIdx.x, threadIdx.x);
}

int main() {
    // Launch CUDA kernel with 1 block and 10 threads
    helloCUDA<<<1, 10>>>();

    // Wait for GPU to finish before accessing stdout
    hipDeviceSynchronize();

    std::cout << "CUDA Kernel Execution Complete!" << std::endl;
    return 0;
}
